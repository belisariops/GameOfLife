

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <iostream>

__device__ int mod(int a, int b) {
    return a >= 0 ? a%b :  ( b - abs ( a%b ) ) % b;
}


__global__ void update(int *A, int *B, int height, int width) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < height*width) {
        int aliveNeighbours = 0;

        int h, w;
        h = index / width;
        w = index - (h * width);

        /*The neighbours of the cell are checked*/
        for (int i = -1; i < 2; ++i) {
            for (int j = -1; j < 2; ++j) {
                aliveNeighbours +=  ( (i | j) && A[mod(h+i,height)*width + mod(w+j,width)]);
            }
        }
        B[index] = (A[index] == 0 && (aliveNeighbours == 3 || aliveNeighbours == 6))
                   || A[index] == 1 && (aliveNeighbours == 2 || aliveNeighbours == 3);
    }
}

void updateCuda(int *A, int *B, int height, int width) {
    // Device input vectors
    int *d_a;
    //Device output vector
    int *d_b;

    // Size, in bytes, of each vector
    size_t bytes = height*width*sizeof(int);

    // Allocate memory for each vector on GPU
    hipMalloc((void **) &d_a, bytes);
    hipMalloc((void **) &d_b, bytes);

    // Copy host vectors to device
    hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);


    int blockSize, gridSize, n;

//    // Tamaño de la matriz.
//    n = height*width;
//
//    // Tamaño del bloque. Elegir entre 32 y 31.
//    //blockSize = 32;
//    blockSize = 32;
//
//    // Number of thread blocks in grid
//    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    update<<< width*height, 1 >>>(d_a, d_b, height, width);

    // Copy array back to host
    hipMemcpy( B, d_b, bytes, hipMemcpyDeviceToHost );



    // Release device memory
    hipFree(d_a);
    hipFree(d_b);

}