


#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
// Device input vectors
int *d_a;
//Device output vector
int *d_b;

__device__ int mod(int a, int b) {
    return a >= 0 ? a%b :  ( b - abs ( a%b ) ) % b;
}


__global__ void update(int *A, int *B, int height, int width) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < height*width) {
        int aliveNeighbours = 0;


        const int h = index / width;
        const int w = index % width;

        /*The neighbours of the cell are checked*/
        for (int i = -1; i < 2; ++i) {
            for (int j = -1; j < 2; ++j) {
                aliveNeighbours +=  ( (i | j) && A[mod(h+i,height)*width + mod(w+j,width)]);
            }
        }
        B[index] = (A[index] == 0 && (aliveNeighbours == 3 || aliveNeighbours == 6))
                   || A[index] == 1 && (aliveNeighbours == 2 || aliveNeighbours == 3);
    }
}

void setUp(int height, int width) {
    // Allocate memory for each vector on GPU
    hipMalloc((void **) &d_a, width*height);
    hipMalloc((void **) &d_b, width*height);
}

void destroy() {
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
}

void updateCuda(int *A, int *B, int height, int width) {


    // Size, in bytes, of each vector
    size_t bytes = height*width*sizeof(int);


    // Copy host vectors to device
    hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);


    int blockSize, gridSize, n;

//    // Tamaño de la matriz.
//    n = height*width;
//
//    // Tamaño del bloque. Elegir entre 32 y 31.
//    //blockSize = 32;
//    blockSize = 32;
//
//    // Number of thread blocks in grid
//    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    update<<< width*height, 1 >>>(d_a, d_b, height, width);

    // Copy array back to host
    hipMemcpy( B, d_b, bytes, hipMemcpyDeviceToHost );




}