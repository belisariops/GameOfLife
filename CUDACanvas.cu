

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <iostream>


__global__ void update(int *A, int *B, int h, int w) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < h*w)
        B[index] = A[index];
}

void updateCuda(int *A, int *B, int height, int width) {
    // Device input vectors
    int *d_a;
    //Device output vector
    int *d_b;

    // Size, in bytes, of each vector
    size_t bytes = height*width*sizeof(int);


    // Allocate memory for each vector on GPU
    hipMalloc((void **) &d_a, bytes);
    hipMalloc((void **) &d_b, bytes);

    // Copy host vectors to device
    hipMemcpy(d_a, A, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize, n;

    // Tamaño de la matriz.
    n = height*width;

    // Tamaño del bloque. Elegir entre 32 y 31.
    //blockSize = 32;
    blockSize = 32;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    update<<< gridSize, blockSize >>>(d_b, d_a, height, width);

    // Copy array back to host
    hipMemcpy( B, d_b, bytes, hipMemcpyDeviceToHost );



    // Release device memory
    hipFree(d_a);
    hipFree(d_b);


}